#include "hip/hip_runtime.h"
// softmax_kernel.cu
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

__global__ void softmax_kernel(float* input, float* output, int num_rows, int num_cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float max_val = -1e20;
        // 找到每行的最大值
        for (int i = 0; i < num_cols; ++i) {
            max_val = fmaxf(max_val, input[row * num_cols + i]);
        }

        // 计算指数和
        float sum_exp = 0.0;
        for (int i = 0; i < num_cols; ++i) {
            sum_exp += expf(input[row * num_cols + i] - max_val);
        }

        // 计算softmax输出
        for (int i = 0; i < num_cols; ++i) {
            output[row * num_cols + i] = expf(input[row * num_cols + i] - max_val) / sum_exp;
        }
    }
}

// Wrapper function
void softmax(torch::Tensor input, torch::Tensor output) {
    const auto num_rows = input.size(0);
    const auto num_cols = input.size(1);

    // 检查输入张量是否在 CUDA 上
    if (!input.is_cuda() || !output.is_cuda()) {
        throw std::invalid_argument("Input and output tensors must be on CUDA device");
    }

    // 检查输入和输出张量的大小
    TORCH_CHECK(input.dim() == 2, "Input tensor must be 2-dimensional");
    TORCH_CHECK(output.dim() == 2, "Output tensor must be 2-dimensional");
    TORCH_CHECK(input.size(0) == output.size(0) && input.size(1) == output.size(1),
                "Input and output tensors must have the same shape");

    // 设置 CUDA 网格和块尺寸
    const int threads_per_block = 256;
    const int blocks_per_grid = (num_rows + threads_per_block - 1) / threads_per_block;

    // 启动 CUDA 内核
    softmax_kernel<<<blocks_per_grid, threads_per_block>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        num_rows,
        num_cols
    );
}
